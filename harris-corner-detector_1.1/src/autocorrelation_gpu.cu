#include "hip/hip_runtime.h"
#include "autocorrelation_gpu.cuh"

__global__
void compute_autocorrelation_matrix_kernel(float *Ix, float *Iy, float *A, float *B, float *C, int nx, int ny) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int p = i * nx + j;
    A[p] = Ix[p]*Ix[p];
    B[p] = Ix[p]*Iy[p];
    C[p] = Iy[p]*Iy[p];
}

void compute_autocorrelation_matrix_cuda(float *Ix, float *Iy, float *A, float *B, float *C, int nx, int ny){
    int imageSize = nx * ny;
    // Allocate device memory
    float *d_Ix, *d_Iy, *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_Ix, imageSize * sizeof(float));
    hipMalloc((void**)&d_Iy, imageSize * sizeof(float));
    hipMalloc((void**)&d_A, imageSize * sizeof(float));
    hipMalloc((void**)&d_B, imageSize * sizeof(float));
    hipMalloc((void**)&d_C, imageSize * sizeof(float));
    // copy in the memory
    hipMemcpy(Ix, d_Ix, imageSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Iy, d_Iy, imageSize * sizeof(float), hipMemcpyDeviceToHost);
    // Define block and grid dimensions
    dim3 blockSize(16, 16); // Adjust block size based on your GPU architecture
    dim3 gridSize(nx / blockSize.x, ny / blockSize.y);
    // Launch the kernel
    compute_autocorrelation_matrix_kernel<<<gridSize, blockSize>>>(d_Ix, d_Iy, d_A, d_B, d_C, nx, ny);
    // Wait for kernel completion
    hipDeviceSynchronize();
    // Copy the results back to host
    hipMemcpy(A, d_A, imageSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(B, d_B, imageSize * sizeof(float), hipMemcpyDeviceToHost);    
    hipMemcpy(C, d_C, imageSize * sizeof(float), hipMemcpyDeviceToHost);    
    // Free device memory
    hipFree(d_Ix);
    hipFree(d_Iy);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
