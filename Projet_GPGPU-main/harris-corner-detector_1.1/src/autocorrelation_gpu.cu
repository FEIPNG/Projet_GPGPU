#include "hip/hip_runtime.h"
#include "autocorrelation_gpu.cuh"
#include <stdio.h>
__global__ void compute_autocorrelation_matrix_kernel(float *Ix, float *Iy, float *A, float *B, float *C, int nx, int ny) {
    int x = blockIdx.x * blockDim.x + threadIdx.x ;
    int y = blockIdx.y * blockDim.y + threadIdx.y ;
    int p = x + y*blockDim.x*gridDim.x ;
    A[p] = Ix[p]*Ix[p];
    B[p] = Ix[p]*Iy[p];
    C[p] = Iy[p]*Iy[p];
    if(p == 0)
        printf("%f, %f ",Ix[0], A[0]);
}

void compute_autocorrelation_matrix_cuda(float *Ix, float *Iy, float *A, float *B, float *C, int nx, int ny){
    int imageSize = nx * ny;
    // Allocate device memory
    float *d_Ix, *d_Iy, *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_Ix, imageSize * sizeof(float));
    hipMalloc((void**)&d_Iy, imageSize * sizeof(float));
    hipMalloc((void**)&d_A, imageSize * sizeof(float));
    hipMalloc((void**)&d_B, imageSize * sizeof(float));
    hipMalloc((void**)&d_C, imageSize * sizeof(float));
    
    // copy in the memory
    hipMemcpy(d_Ix, Ix, imageSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Iy, Iy, imageSize * sizeof(float), hipMemcpyHostToDevice);
    // Define block and grid dimensions
    dim3 blockSize(10, 10); // Adjust block size based on your GPU architecture
    dim3 gridSize((int)(nx/blockSize.x), (int)(ny / blockSize.y));
    // Launch the kernel
    compute_autocorrelation_matrix_kernel<<<gridSize, blockSize>>>(d_Ix, d_Iy, d_A, d_B, d_C, nx, ny);
    // Wait for kernel completion
    hipDeviceSynchronize();
    // Copy the results back to host
    hipMemcpy(A, d_A, imageSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(B, d_B, imageSize * sizeof(float), hipMemcpyDeviceToHost);    
    hipMemcpy(C, d_C, imageSize * sizeof(float), hipMemcpyDeviceToHost);
    // printf("A:\n");
    // for (int j = 0; j < 100; j++){
	// 	printf("%f",A[j]);
	// }
    // printf("B:\n");
    // for (int j = 0; j < imageSize; j++){
	// 	printf("%f",B[j]);
	// }
    // printf("C:\n");
    // for (int j = 0; j < imageSize; j++){
	// 	printf("%f",C[j]);
	// }
    // Free device memory
    hipFree(d_Ix);
    hipFree(d_Iy);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
