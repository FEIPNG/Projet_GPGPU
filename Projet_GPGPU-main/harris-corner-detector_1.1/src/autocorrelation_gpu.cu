#include "hip/hip_runtime.h"
#include "autocorrelation_gpu.cuh"
#include <stdio.h>

texture<float, 1, hipReadModeElementType> texIx;
texture<float, 1, hipReadModeElementType> texIy;

__global__ void compute_autocorrelation_matrix_kernel_texture(float *A, float *B, float *C) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = x + y * blockDim.x * gridDim.x;

    float Ix_value = tex1D(texIx, p);  // Read from texture memory
    float Iy_value = tex1D(texIy, p);

    A[p] = Ix_value * Ix_value;
    B[p] = Ix_value * Iy_value;
    C[p] = Iy_value * Iy_value;

    if (p == 0)
        printf("%f, %f ", Ix_value, A[0]);
}

// __global__ void compute_autocorrelation_matrix_kernel(float *Ix, float *Iy, float *A, float *B, float *C, int nx, int ny) {
//     int x = blockIdx.x * blockDim.x + threadIdx.x ;
//     int y = blockIdx.y * blockDim.y + threadIdx.y ;
//     int p = x + y*blockDim.x*gridDim.x ;
//     A[p] = Ix[p]*Ix[p];
//     B[p] = Ix[p]*Iy[p];
//     C[p] = Iy[p]*Iy[p];
//     if(p == 0)
//         printf("%f, %f ",Ix[0], A[0]);
// }

//avec texture
void compute_autocorrelation_matrix_cuda(float *Ix, float *Iy, float *A, float *B, float *C, int nx, int ny){
    int imageSize = nx * ny;
    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Texture
    hipArray *d_IxArray, *d_IyArray;
    hipMallocArray(&d_IxArray, &texIx.channelDesc, imageSize, 1);
    hipMallocArray(&d_IyArray, &texIy.channelDesc, imageSize, 1);
    // Copy input data to texture memory
    hipMemcpyToArray(d_IxArray, 0, 0, Ix, imageSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToArray(d_IyArray, 0, 0, Iy, imageSize * sizeof(float), hipMemcpyHostToDevice);

    // Bind texture references to the arrays
    hipBindTextureToArray(texIx, d_IxArray);
    hipBindTextureToArray(texIy, d_IyArray);

    hipMalloc((void**)&d_A, imageSize * sizeof(float));
    hipMalloc((void**)&d_B, imageSize * sizeof(float));
    hipMalloc((void**)&d_C, imageSize * sizeof(float));
    
    // Define block and grid dimensions
    dim3 blockSize(10, 10); // Adjust block size based on your GPU architecture
    dim3 gridSize((int)(nx/blockSize.x), (int)(ny / blockSize.y));
    // Launch the kernel
    compute_autocorrelation_matrix_kernel_texture<<<gridSize, blockSize>>>(d_A, d_B, d_C);

    // Wait for kernel completion
    hipDeviceSynchronize();
    // Copy the results back to host
    hipMemcpy(A, d_A, imageSize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(B, d_B, imageSize * sizeof(float), hipMemcpyDeviceToHost);    
    hipMemcpy(C, d_C, imageSize * sizeof(float), hipMemcpyDeviceToHost);

    // Unbind texture references
    hipUnbindTexture(texIx);
    hipUnbindTexture(texIy);
    // printf("A:\n");
    // for (int j = 0; j < 100; j++){
	// 	printf("%f",A[j]);
	// }
    // printf("B:\n");
    // for (int j = 0; j < imageSize; j++){
	// 	printf("%f",B[j]);
	// }
    // printf("C:\n");
    // for (int j = 0; j < imageSize; j++){
	// 	printf("%f",C[j]);
	// }
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFreeArray(d_IxArray);
    hipFreeArray(d_IyArray);
}

// sans texture 
// void compute_autocorrelation_matrix_cuda(float *Ix, float *Iy, float *A, float *B, float *C, int nx, int ny){
//     int imageSize = nx * ny;
//     // Allocate device memory
//     float *d_Ix, *d_Iy, *d_A, *d_B, *d_C;

//     hipMalloc((void**)&d_Ix, imageSize * sizeof(float));
//     hipMalloc((void**)&d_Iy, imageSize * sizeof(float));
//     hipMalloc((void**)&d_A, imageSize * sizeof(float));
//     hipMalloc((void**)&d_B, imageSize * sizeof(float));
//     hipMalloc((void**)&d_C, imageSize * sizeof(float));
    
//     // copy in the memory
//     hipMemcpy(d_Ix, Ix, imageSize * sizeof(float), hipMemcpyHostToDevice);
//     hipMemcpy(d_Iy, Iy, imageSize * sizeof(float), hipMemcpyHostToDevice);
//     // Define block and grid dimensions
//     dim3 blockSize(10, 10); // Adjust block size based on your GPU architecture
//     dim3 gridSize((int)(nx/blockSize.x), (int)(ny / blockSize.y));
//     // Launch the kernel
//     compute_autocorrelation_matrix_kernel<<<gridSize, blockSize>>>(d_Ix, d_Iy, d_A, d_B, d_C, nx, ny);
//     // Wait for kernel completion
//     hipDeviceSynchronize();
//     // Copy the results back to host
//     hipMemcpy(A, d_A, imageSize * sizeof(float), hipMemcpyDeviceToHost);
//     hipMemcpy(B, d_B, imageSize * sizeof(float), hipMemcpyDeviceToHost);    
//     hipMemcpy(C, d_C, imageSize * sizeof(float), hipMemcpyDeviceToHost);
//     // printf("A:\n");
//     // for (int j = 0; j < 100; j++){
// 	// 	printf("%f",A[j]);
// 	// }
//     // printf("B:\n");
//     // for (int j = 0; j < imageSize; j++){
// 	// 	printf("%f",B[j]);
// 	// }
//     // printf("C:\n");
//     // for (int j = 0; j < imageSize; j++){
// 	// 	printf("%f",C[j]);
// 	// }
//     // Free device memory
//     hipFree(d_Ix);
//     hipFree(d_Iy);
//     hipFree(d_A);
//     hipFree(d_B);
//     hipFree(d_C);
// }
